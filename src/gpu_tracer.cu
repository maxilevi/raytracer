#include "hip/hip_runtime.h"
/*
 * Created by Maximiliano Levi on 3/19/2021.
 */

#include "gpu_tracer.h"
#include "defines.h"
#include "math/ray.h"
#include "camera.h"

#define THREAD_COUNT 1024
#define PI 3.14159265
#define MAX_DOUBLE DBL_MAX
#define MIN(x, y) x < y ? x : y

CUDA_DEVICE double RandomDouble(uint32_t& seed)
{
    int k;
    int s = int(seed);
    if (s == 0)
        s = 305420679;
    k = s / 127773;
    s = 16807 * (s - k * 127773) - 2836 * k;
    if (s < 0)
        s += 2147483647;
    seed = uint32_t(s);
    return double(seed % uint32_t(65536)) / 65535.0;
}

CUDA_DEVICE Vector3 RandomPointOnUnitSphere(uint32_t& seed)
{
    double u1 = RandomDouble(seed);
    double u2 = RandomDouble(seed);
    double lambda = acos(2.0 * u1 - 1) - PI / 2.0;
    double phi = 2.0 * PI * u2;
    return {std::cos(lambda) * std::cos(phi), std::cos(lambda) * std::sin(phi), std::sin(lambda)};
}

CUDA_DEVICE Vector3 BackgroundColor(const Ray& ray)
{
    auto unit_dir = Vector3(ray.Direction()).Normalized();
    double t = 0.5 * (unit_dir.Y() + 1.0);
    return (1.0 - t) * Vector3(1) + t * Vector3(0.5, 0.7, 1.0);
}

CUDA_DEVICE Vector3 Color(Scene& scene, const Ray& ray, uint32_t& seed)
{
    Ray current_ray = ray;
    HitResult result;
    Vector3 color = Vector3(1);
    int iteration = 0;
    while (scene.Hit(current_ray, 0.001, MAX_DOUBLE, result))
    {
        Vector3 target_direction = result.Normal + RandomPointOnUnitSphere(seed);
        current_ray = Ray(result.Point, target_direction);
        color *= 0.5;
        if (iteration++ == Camera::kMaxLightBounces)
            return {0, 0, 0};
    }
    return color * BackgroundColor(current_ray);
}

__global__
void ColorKernel(Scene scene, Vector3* out_colors, const int* device_params, int n, int width, int height, Vector3 origin, Vector3 screen, Vector3 step_x, Vector3 step_y)
{
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    uint32_t seed = idx;
    int i = device_params[2*idx];
    int j = device_params[2*idx+1];
    double noise = RandomDouble(seed);
    double u = (i + noise) / double(width);
    double v = (j + noise) / double(height);

    Ray r(origin, screen + step_x * u + step_y * v);
    out_colors[idx] = Color(scene, r, seed);
}

void GPUTrace(Scene& scene, const std::vector<std::pair<int, int>>& params, Vector3* colors, int width, int height)
{
    double screen_ratio = (double(width) / double(height));
    Vector3 origin(0, 0, 0);
    Vector3 screen(-screen_ratio, -1, -1);
    Vector3 step_x(std::abs(screen_ratio) * 2.0, 0, 0);
    Vector3 step_y(0, 2, 0);

    int n = params.size();
    int step = n / 2;

    std::cout << "Total samples to process = " << n << std::endl;

    Vector3* out_colors;
    int* device_params;
    auto* all_samples = new Vector3[step];

    CUDA_CALL(hipMalloc(&device_params, step * 2 * sizeof(int)));
    CUDA_CALL(hipMalloc(&out_colors, step * sizeof(Vector3)));

    for(size_t w = 0; w < n; w += step)
    {
        std::cout << "Processing elements (" << w << ", " << w + step << ")" << std::endl;
        int size = MIN(step, params.size() - w);
        int blocks = (size + THREAD_COUNT - 1) / THREAD_COUNT;
        std::cout << "Launching " << blocks << " blocks with " << THREAD_COUNT << " threads each (" << (blocks * THREAD_COUNT) << " total threads) for " << step << " elements" << std::endl;

        CUDA_CALL(hipMemcpy(device_params, &params[0] + w, size * 2 * sizeof(int), hipMemcpyHostToDevice));

        ColorKernel<<<blocks, THREAD_COUNT>>>(scene, out_colors, device_params, size, width, height, origin, screen, step_x, step_y);

        CUDA_CALL(hipPeekAtLastError())
        CUDA_CALL(hipDeviceSynchronize());

        std::cout << "All CUDA threads joined." << std::endl;

        std::cout << "Copying CUDA results." << std::endl;

        CUDA_CALL(hipMemcpy(all_samples, out_colors, size * sizeof(Vector3), hipMemcpyDeviceToHost));

        std::cout << "Collapsing samples..." << std::endl;

        /* Reduce all samples into one */
        for (size_t k = 0; k < size; ++k) {
            int i = params[k + w].first;
            int j = params[k + w].second;
            auto color = all_samples[k];
            colors[j * width + i] += color;
        }
    }

    delete[] all_samples;
    CUDA_CALL(hipFree(out_colors));
    CUDA_CALL(hipFree(device_params));

    std::cout << "Finished CUDA work." << std::endl;
}

__global__ void BvhKernel()
{

}

void BuildBvh()
{
    BvhKernel<<1, 1>>();
}