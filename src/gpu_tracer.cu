#include "hip/hip_runtime.h"
/*
 * Created by Maximiliano Levi on 3/19/2021.
 */

#include "gpu_tracer.h"
#include "kernel/helper.h"
#include "math/ray.h"
#include "camera.h"
#include "volumes/bvh.h"
#include "kernel/random.h"
#include "volumes/gpu_bvh.h"

#define THREAD_COUNT 256

CUDA_DEVICE Vector3 RandomPointOnUnitSphere(uint32_t& seed)
{
    double u1 = RandomDouble(seed);
    double u2 = RandomDouble(seed);
    double lambda = acos(2.0 * u1 - 1) - PI / 2.0;
    double phi = 2.0 * PI * u2;
    return {std::cos(lambda) * std::cos(phi), std::cos(lambda) * std::sin(phi), std::sin(lambda)};
}

CUDA_DEVICE Vector3 BackgroundColor(const Ray& ray)
{
    auto unit_dir = Vector3(ray.Direction()).Normalized();
    double t = 0.5 * (unit_dir.Y() + 1.0);
    return (1.0 - t) * Vector3(1) + t * Vector3(0.5, 0.7, 1.0);
}

CUDA_DEVICE Vector3 Color(GPUBvh& bvh, const Ray& ray, uint32_t& seed)
{
    Ray current_ray = ray;
    HitResult result;
    Vector3 color = Vector3(1);
    int iteration = 0;
    while (bvh.Hit(current_ray, 0.001, MAX_DOUBLE, result))
    {
        Vector3 target_direction = result.Normal + RandomPointOnUnitSphere(seed);
        current_ray = Ray(result.Point, target_direction);
        color *= 0.5;
        if (iteration++ == Camera::kMaxLightBounces)
            return {0, 0, 0};
    }
    return color * BackgroundColor(current_ray);
}

__global__
void ColorKernel(GPUBvh bvh, Vector3* out_colors, const int* device_params, int n, int width, int height, Vector3 origin, Vector3 screen, Vector3 step_x, Vector3 step_y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    uint32_t seed = idx;
    int i = device_params[2*idx];
    int j = device_params[2*idx+1];
    double noise = RandomDouble(seed);
    double u = (i + noise) / double(width);
    double v = (j + noise) / double(height);

    Ray r(origin, screen + step_x * u + step_y * v);
    out_colors[idx] = Color(bvh, r, seed);
}

void GPUTrace(Scene& scene, const std::vector<std::pair<int, int>>& params, Vector3* colors, int width, int height)
{
    double screen_ratio = (double(width) / double(height));
    Vector3 origin(0, 0, 0);
    Vector3 screen(-screen_ratio, -1, -1);
    Vector3 step_x(std::abs(screen_ratio) * 2.0, 0, 0);
    Vector3 step_y(0, 2, 0);

    int n = params.size();
    int step = n / 2;

    std::cout << "Total samples to process = " << n << std::endl;

    Vector3* out_colors;
    int* device_params;
    auto* all_samples = new Vector3[step];

    CUDA_CALL(hipMalloc(&device_params, step * 2 * sizeof(int)));
    CUDA_CALL(hipMalloc(&out_colors, step * sizeof(Vector3)));

    std::cout << "Generating GPU Bvh" << std::endl;

    GPUBvh gpu_bvh = GPUBvh::FromBvh(scene.GetBvh());

    std::cout << "Starting CUDA work" << std::endl;

    for(size_t w = 0; w < n; w += step)
    {
        std::cout << "Processing elements (" << w << ", " << w + step << ")" << std::endl;
        int size = MIN(step, params.size() - w);
        int blocks = (size + THREAD_COUNT - 1) / THREAD_COUNT;
        std::cout << "Launching " << blocks << " blocks with " << THREAD_COUNT << " threads each (" << (blocks * THREAD_COUNT) << " total threads) for " << step << " elements" << std::endl;

        CUDA_CALL(hipMemcpy(device_params, &params[0] + w, size * 2 * sizeof(int), hipMemcpyHostToDevice));

        ColorKernel<<<blocks, THREAD_COUNT>>>(gpu_bvh, out_colors, device_params, size, width, height, origin, screen, step_x, step_y);

        CUDA_CALL(hipPeekAtLastError())
        CUDA_CALL(hipDeviceSynchronize());

        std::cout << "All CUDA threads joined." << std::endl;

        std::cout << "Copying CUDA results." << std::endl;

        CUDA_CALL(hipMemcpy(all_samples, out_colors, size * sizeof(Vector3), hipMemcpyDeviceToHost));

        std::cout << "Collapsing samples..." << std::endl;

        /* Reduce all samples into one */
        for (size_t k = 0; k < size; ++k) {
            int i = params[k + w].first;
            int j = params[k + w].second;
            auto color = all_samples[k];
            colors[j * width + i] += color;
        }
    }

    delete[] all_samples;
    CUDA_CALL(hipFree(out_colors));
    CUDA_CALL(hipFree(device_params));
    GPUBvh::Delete(gpu_bvh);

    std::cout << "Finished CUDA work." << std::endl;
}