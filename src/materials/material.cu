/*
 * Created by Maximiliano Levi on 4/14/2021.
 */

#include "material.h"
#include "../io/tga.h"
#include "../io/stb_image.h"
#include <string>
#include <assert.h>

int Material::ID_COUNTER = 0;

Material::Material(const char *filename)
{
    int w, h, n;
    unsigned char *data = stbi_load(filename, &w, &h, &n, 3);
    assert(w != 0);
    assert(h != 0);
    assert(n == 3);
    auto colors = new uint8_t[w * h * 3];
    for(size_t j = 0; j < h * w * 3; ++j)
        colors[j] = data[j];

    stbi_image_free(data);
    this->texture_ = colors;
    this->width_ = w;
    this->height_ = h;
    this->texel_width_ = (1.0 / width_);
    this->texel_height_ = (1.0 / height_);
    this->is_in_gpu_ = false;
    this->id_ = ID_COUNTER++;
}

CUDA_HOST_DEVICE Vector3 Material::Sample(double s, double t) const
{
    if (s < 0.0 || s > 1.0)
        printf("error %f", s);
    if (t < 0.0 || t > 1.0)
        printf("error %f", t);
    auto x = size_t(s * width_);
    auto y = size_t(t * height_);

    Vector3 ans;
    for(auto i = 0; i < 3; ++i)
    {
        //Modulo(ref X) * _boundsY * _boundsZ + Y * _boundsZ + Modulo(ref Z)
        auto idx = y * width_ * 3 + x * 3 + i;//y * width_ * 3 + x * 3 + i;
        ans[i] = this->texture_[idx];
    }
    return ans / 256.0;
    //printf("%d, %d %d\n", (int)offset, (int)width_, (int)height_);
    //return Vector3(this->texture_[offset + 0], this->texture_[offset + 1], this->texture_[offset + 2]) / 256.0;
}

CUDA_HOST_DEVICE Vector3 Material::BilinearSample(double s, double t) const
{
    auto floored_s = (size_t(s * width_) / double(width_));
    auto floored_t = (size_t(t * height_) / double(height_));
    auto x = (s - floored_s) / texel_width_;
    auto y = (t - floored_t) / texel_height_;

    auto center = this->Sample(s, t);
    auto top_left = this->Sample(s - texel_width_, t + texel_height_);
    auto bot_left = this->Sample(s - texel_width_, t - texel_height_);
    auto top_right = this->Sample(s + texel_width_, t + texel_height_);
    auto bot_right = this->Sample(s + texel_width_, t - texel_height_);

    return Vector3::Lerp(Vector3::Lerp(bot_left, bot_right, x), Vector3::Lerp(top_left, top_right, x), y);
}

CUDA_HOST_DEVICE Material::~Material()
{
    if (!is_in_gpu_)
        delete[] texture_;
}

Material Material::MakeGPUMaterial()
{
    Material mat;
    mat.is_in_gpu_ = true;
    mat.texel_height_ = texel_height_;
    mat.texel_width_ = texel_width_;
    mat.width_ = width_;
    mat.height_ = height_;
    mat.id_ = id_;
    assert(mat.width_ != 0);

    CUDA_CALL(hipMalloc(&mat.texture_, sizeof(uint8_t) * width_ * height_ * 3));
    CUDA_CALL(hipMemcpy(mat.texture_, texture_, sizeof(uint8_t) * width_ * height_ * 3, hipMemcpyHostToDevice));
    return mat;
}

void Material::FreeGPUMaterial()
{
    CUDA_CALL(hipFree(texture_));
}

int Material::Id()
{
    return id_;
}
