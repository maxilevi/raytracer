/*
 * Created by Maximiliano Levi on 4/14/2021.
 */

#include "material.h"
#include "../io/tga.h"
#include "../io/stb_image.h"
#include <string>
#include <assert.h>

int Material::ID_COUNTER = 0;

Material::Material(const char *filename)
{
    int w, h, n;
    uchar_t *data = stbi_load(filename, &w, &h, &n, 3);
    assert(w != 0);
    assert(h != 0);
    assert(n == 3);
    auto colors = new uchar_t[w * h * 3];
    for(size_t j = 0; j < h * w * 3; ++j)
        colors[j] = data[j];

    stbi_image_free(data);
    this->texture_ = colors;
    this->width_ = w;
    this->height_ = h;
    this->texel_width_ = (1.0 / width_);
    this->texel_height_ = (1.0 / height_);
    this->is_in_gpu_ = false;
    this->id_ = ID_COUNTER++;
}

CUDA_HOST_DEVICE Vector3 Material::Sample(double s, double t) const
{
    auto x = size_t(s * width_);
    auto y = size_t(t * height_);

    Vector3 ans;
    for(auto i = 0; i < 3; ++i)
    {
        auto idx = y * width_ * 3 + x * 3 + i;
        ans[i] = this->texture_[idx];
    }
    //if (ans.X() <= DBL_EPSILON && ans.Y() <= DBL_EPSILON && ans.Z() <= DBL_EPSILON)
    //    printf("Pixels %d %d have no color\n", (int)x, (int)y);
    return ans / 256.0;
}

CUDA_HOST_DEVICE Vector3 Material::BilinearSample(double s, double t) const
{
    auto floored_s = (size_t(s * width_) / double(width_));
    auto floored_t = (size_t(t * height_) / double(height_));
    auto x = (s - floored_s) / texel_width_;
    auto y = (t - floored_t) / texel_height_;
    auto tw = texel_width_;
    auto th = texel_height_;

    auto center = this->Sample(s, t);
    auto top_left = this->Sample(s - tw, t + th);
    auto bot_left = this->Sample(s - tw, t - th);
    auto top_right = this->Sample(s + tw, t + th);
    auto bot_right = this->Sample(s + tw, t - th);

    return Vector3::Lerp(Vector3::Lerp(bot_left, bot_right, x), Vector3::Lerp(top_left, top_right, x), y);
}

CUDA_HOST_DEVICE Material::~Material()
{
    if (!is_in_gpu_)
        delete[] texture_;
}

Material Material::MakeGPUMaterial()
{
    Material mat;
    mat.is_in_gpu_ = true;
    mat.texel_height_ = texel_height_;
    mat.texel_width_ = texel_width_;
    mat.width_ = width_;
    mat.height_ = height_;
    mat.id_ = id_;
    assert(mat.width_ != 0);
    assert(mat.height_ != 0);

    CUDA_CALL(hipMalloc(&mat.texture_, sizeof(uchar_t) * width_ * height_ * 3));
    CUDA_CALL(hipMemcpy(mat.texture_, texture_, sizeof(uchar_t) * width_ * height_ * 3, hipMemcpyHostToDevice));
    return mat;
}

void Material::FreeGPUMaterial()
{
    CUDA_CALL(hipFree(texture_));
}

int Material::Id()
{
    return id_;
}
/*
bool Material::Scatter(const Ray& , const HitResult & result, Vector3 &attenuation, Ray &) const
{
    auto scatter_direction = rec.normal + random_unit_vector();
    scattered = ray(rec.p, scatter_direction);
    attenuation = albedo;
    return true;

    Vector3 reflected = reflect(unit_vector(r_in.direction()), rec.normal);
    scattered = ray(rec.p, reflected);
    attenuation = albedo;
    return (dot(scattered.direction(), rec.normal) > 0);
}
*/